#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100000000
#define BLOCK_SIZE 256

// CUDA kernel
__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    size_t size = N * sizeof(float);

    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = (float)i;
        h_b[i] = (float)i;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Timing using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipEventRecord(start);
    vectorAdd<<<numBlocks, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);

    // Wait for GPU to finish
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print first 10 results
    printf("Result (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_c[i]);
    }
    printf("\n");

    printf("Kernel execution time: %.3f ms\n", milliseconds);

    // Free memory
    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
